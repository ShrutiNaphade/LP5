
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

using namespace std;

__global__ void VectorAdd(int* a,int* b,int* c,int size){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<size){
		c[tid]=a[tid]+b[tid];
	}
}

int main(){
	
	int size=500;
	int numbytes=size*sizeof(int);
	
	int* h_a=(int*)malloc(numbytes);
	int* h_b=(int*)malloc(numbytes);
	int* h_c=(int*)malloc(numbytes);
	
	for(int i=0;i<size;i++){
		h_a[i]=i;
		h_b[i]=size-i;
	}
	
	int* d_a;
	int* d_b;
	int* d_c;
	hipMalloc((void**)&d_a,numbytes);
	hipMalloc((void**)&d_b,numbytes);
	hipMalloc((void**)&d_c,numbytes);
	
	hipMemcpy(d_a,h_a,numbytes,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,numbytes,hipMemcpyHostToDevice);
	
	int blocksize=256;
	int gridsize=(size+blocksize-1)/blocksize;
	
	VectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,size);
	
	hipMemcpy(h_c,d_c,numbytes,hipMemcpyDeviceToHost);
	
	for(int i=0;i<size;i++){
		cout<<"\n"<<h_a[i]<<"+"<<h_b[i]<<"="<<h_c[i];
	}
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	free(h_a);
	free(h_b);
	free(h_c);
	
	
	return 0;
}
